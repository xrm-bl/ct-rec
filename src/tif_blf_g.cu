#include "hip/hip_runtime.h"
/*
 * 3D Bilateral Filter for Large TIFF Image Stacks - CUDA Streaming Version
 * Handles datasets larger than GPU memory by processing in chunks
 * Optimized for maximum GPU utilization with minimal memory usage
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifdef _WIN32
    #define WIN32_LEAN_AND_MEAN
    #include <windows.h>
    #include <direct.h>
    #include "tiffio.h"
    #define PATH_SEPARATOR "\\"
    #define mkdir(path, mode) _mkdir(path)
    #define snprintf _snprintf
    #ifndef isfinite
        #define isfinite(x) _finite(x)
    #endif
#else
    #include <unistd.h>
    #include <dirent.h>
    #include <sys/stat.h>
    #include <sys/sysinfo.h>
    #include <tiffio.h>
    #define PATH_SEPARATOR "/"
#endif

/* CUDA error checking macro */
#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
} while(0)

/* Constants */
#define MAX_PATH_LENGTH 1024
#define MAX_FILES 10000
#define LOG_BUFFER_SIZE 4096
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 4
#define MAX_KERNEL_SIZE 21
#define GPU_MEMORY_FRACTION 0.8f  /* Use 80% of GPU memory */

/* Structure for filter parameters */
typedef struct {
    int kernel_size;
    float spatial_sigma;
    float intensity_sigma;
} FilterParams;

/* Structure for image metadata */
typedef struct {
    unsigned int width;
    unsigned int height;
    unsigned int depth;
    unsigned short bits_per_sample;
    unsigned short samples_per_pixel;
    unsigned short sample_format;
    size_t bytes_per_pixel;
    size_t bytes_per_slice;
} ImageInfo;

/* Structure for GPU chunk processing */
typedef struct {
    int start_z;
    int end_z;
    int chunk_depth;
    void *h_data;      /* Host pinned memory */
    void *d_data;      /* Device memory */
    void *d_output;    /* Device output memory */
    hipStream_t stream;
} GPUChunk;

/* Global variables for logging */
static FILE *log_file = NULL;

/* CUDA kernel for 3D bilateral filter - optimized for streaming */
template<typename T>
__global__ void bilateral_filter_3d_kernel_stream(
    const T* __restrict__ input,
    T* __restrict__ output,
    int width, int height, int chunk_depth,
    int kernel_size, 
    float spatial_sigma_sq_inv,
    float intensity_sigma_sq_inv,
    float max_value,
    int global_z_offset,
    int total_depth,
    const T* __restrict__ prev_overlap,
    const T* __restrict__ next_overlap,
    int prev_overlap_depth,
    int next_overlap_depth)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x >= width || y >= height || z >= chunk_depth) return;
    
    const int half_kernel = kernel_size / 2;
    const size_t slice_size = (size_t)width * height;
    const size_t center_idx = (size_t)z * slice_size + (size_t)y * width + x;
    const float center_value = (float)input[center_idx];
    
    float weighted_sum = 0.0f;
    float weight_sum = 0.0f;
    
    /* Process kernel */
    for (int kz = -half_kernel; kz <= half_kernel; kz++) {
        const int local_z = z + kz;
        const int global_z = global_z_offset + z + kz;
        
        /* Skip if outside global bounds */
        if (global_z < 0 || global_z >= total_depth) continue;
        
        for (int ky = -half_kernel; ky <= half_kernel; ky++) {
            const int ny = y + ky;
            if (ny < 0 || ny >= height) continue;
            
            for (int kx = -half_kernel; kx <= half_kernel; kx++) {
                const int nx = x + kx;
                if (nx < 0 || nx >= width) continue;
                
                float neighbor_value;
                
                /* Determine which buffer to read from */
                if (local_z < 0 && prev_overlap != NULL) {
                    /* Read from previous overlap */
                    const int prev_z = prev_overlap_depth + local_z;
                    if (prev_z >= 0) {
                        const size_t prev_idx = (size_t)prev_z * slice_size + 
                                               (size_t)ny * width + nx;
                        neighbor_value = (float)prev_overlap[prev_idx];
                    } else {
                        continue;
                    }
                } else if (local_z >= chunk_depth && next_overlap != NULL) {
                    /* Read from next overlap */
                    const int next_z = local_z - chunk_depth;
                    if (next_z < next_overlap_depth) {
                        const size_t next_idx = (size_t)next_z * slice_size + 
                                               (size_t)ny * width + nx;
                        neighbor_value = (float)next_overlap[next_idx];
                    } else {
                        continue;
                    }
                } else if (local_z >= 0 && local_z < chunk_depth) {
                    /* Read from current chunk */
                    const size_t neighbor_idx = (size_t)local_z * slice_size + 
                                               (size_t)ny * width + nx;
                    neighbor_value = (float)input[neighbor_idx];
                } else {
                    continue;
                }
                
                /* Calculate weights */
                const float spatial_dist_sq = (float)(kx*kx + ky*ky + kz*kz);
                const float spatial_weight = __expf(-spatial_dist_sq * spatial_sigma_sq_inv);
                
                const float intensity_diff = neighbor_value - center_value;
                const float intensity_dist_sq = intensity_diff * intensity_diff;
                const float intensity_weight = __expf(-intensity_dist_sq * intensity_sigma_sq_inv);
                
                const float weight = spatial_weight * intensity_weight;
                weighted_sum += neighbor_value * weight;
                weight_sum += weight;
            }
        }
    }
    
    /* Write output */
    if (weight_sum > 0.0f) {
        float result = weighted_sum / weight_sum;
        result = fminf(fmaxf(result, 0.0f), max_value);
        output[center_idx] = (T)result;
    } else {
        output[center_idx] = input[center_idx];
    }
}

/* Function prototypes */
static void init_logging(const char *program_name);
static void close_logging(void);
static void log_message(const char *format, ...);
static int create_directory(const char *path);
static int get_tiff_files(const char *dir_path, char files[][MAX_PATH_LENGTH], int *file_count);
static int compare_strings(const void *a, const void *b);
static int get_image_info(const char *dir_path, const char *filename, ImageInfo *info);
static size_t get_gpu_memory_available(void);
static int calculate_gpu_chunk_size(ImageInfo *info, int kernel_size);
static GPUChunk* allocate_gpu_chunk(ImageInfo *info, int chunk_depth);
static void free_gpu_chunk(GPUChunk *chunk);
static int load_chunk_to_host(const char *dir_path, char files[][MAX_PATH_LENGTH], 
                              ImageInfo *info, GPUChunk *chunk);
static int save_chunk_from_host(const char *dir_path, char files[][MAX_PATH_LENGTH], 
                                ImageInfo *info, GPUChunk *chunk);
static void process_chunk_on_gpu(GPUChunk *chunk, ImageInfo *info, FilterParams *params,
                                GPUChunk *prev_chunk, GPUChunk *next_chunk);
static double get_current_time(void);
static float get_float_dynamic_range(const char *dir_path, char files[][MAX_PATH_LENGTH], 
                                    int file_count, ImageInfo *info);

/* Initialize logging */
static void init_logging(const char *program_name) {
    /*
    char log_path[MAX_PATH_LENGTH];
    char timestamp[64];
    time_t now;
    struct tm *tm_info;
    const char *home_dir;
    
#ifdef _WIN32
    home_dir = getenv("USERPROFILE");
#else
    home_dir = getenv("HOME");
#endif
    
    if (home_dir == NULL) {
        home_dir = ".";
    }
    
    snprintf(log_path, MAX_PATH_LENGTH, "%s%scom-log", home_dir, PATH_SEPARATOR);
    create_directory(log_path);
    
    time(&now);
    tm_info = localtime(&now);
    strftime(timestamp, sizeof(timestamp), "%Y%m%d_%H%M%S", tm_info);
    
    snprintf(log_path, MAX_PATH_LENGTH, "%s%scom-log%sbilateral_3d_cuda_large_%s.log", 
             home_dir, PATH_SEPARATOR, PATH_SEPARATOR, timestamp);
    
    log_file = fopen(log_path, "w");
    if (log_file == NULL) {
        fprintf(stderr, "Warning: Could not create log file\n");
    }
    
    log_message("=== 3D Bilateral Filter (CUDA Large Data) Started ===");
    log_message("Program: %s", program_name);
    
    // Log CUDA device information
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        log_message("CUDA Device: %s", prop.name);
        log_message("Total Global Memory: %.2f GB", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        log_message("Memory Bus Width: %d bits", prop.memoryBusWidth);
        log_message("Memory Bandwidth: %.2f GB/s", 
                    2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }
    */
}

/* Close logging */
static void close_logging(void) {
    /*
    if (log_file != NULL) {
        log_message("=== 3D Bilateral Filter (CUDA Large Data) Completed ===");
        fclose(log_file);
        log_file = NULL;
    }
    */
}

/* Log message with timestamp */
static void log_message(const char *format, ...) {
/* All logging disabled
    va_list args;
    time_t now;
    struct tm *tm_info;
    char timestamp[32];
    
    if (log_file == NULL) return;
    
    time(&now);
    tm_info = localtime(&now);
    strftime(timestamp, sizeof(timestamp), "%Y-%m-%d %H:%M:%S", tm_info);
    
    fprintf(log_file, "[%s] ", timestamp);
    
    va_start(args, format);
    vfprintf(log_file, format, args);
    va_end(args);
    
    fprintf(log_file, "\n");
    fflush(log_file);
    
    printf("[%s] ", timestamp);
    va_start(args, format);
    vprintf(format, args);
    va_end(args);
    printf("\n");
*/
}

/* Create directory */
static int create_directory(const char *path) {
#ifdef _WIN32
    if (GetFileAttributesA(path) == INVALID_FILE_ATTRIBUTES) {
        if (_mkdir(path) != 0) {
            return -1;
        }
    }
#else
    struct stat st;
    if (stat(path, &st) != 0) {
        if (mkdir(path, 0755) != 0) {
            return -1;
        }
    }
#endif
    return 0;
}

/* String comparison for qsort */
static int compare_strings(const void *a, const void *b) {
    return strcmp((const char *)a, (const char *)b);
}

/* Get list of TIFF files in directory */
static int get_tiff_files(const char *dir_path, char files[][MAX_PATH_LENGTH], int *file_count) {
#ifdef _WIN32
    WIN32_FIND_DATAA find_data;
    HANDLE find_handle;
    char search_path[MAX_PATH_LENGTH];
    
    snprintf(search_path, MAX_PATH_LENGTH, "%s\\*.tif*", dir_path);
    find_handle = FindFirstFileA(search_path, &find_data);
    
    if (find_handle == INVALID_HANDLE_VALUE) {
        return -1;
    }
    
    *file_count = 0;
    do {
        if (!(find_data.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY)) {
            strncpy(files[*file_count], find_data.cFileName, MAX_PATH_LENGTH - 1);
            files[*file_count][MAX_PATH_LENGTH - 1] = '\0';
            (*file_count)++;
            if (*file_count >= MAX_FILES) break;
        }
    } while (FindNextFileA(find_handle, &find_data));
    
    FindClose(find_handle);
#else
    DIR *dir;
    struct dirent *entry;
    char *ext;
    
    dir = opendir(dir_path);
    if (dir == NULL) {
        return -1;
    }
    
    *file_count = 0;
    while ((entry = readdir(dir)) != NULL) {
        ext = strrchr(entry->d_name, '.');
        if (ext != NULL && (strcmp(ext, ".tif") == 0 || strcmp(ext, ".tiff") == 0)) {
            strncpy(files[*file_count], entry->d_name, MAX_PATH_LENGTH - 1);
            files[*file_count][MAX_PATH_LENGTH - 1] = '\0';
            (*file_count)++;
            if (*file_count >= MAX_FILES) break;
        }
    }
    
    closedir(dir);
#endif
    
    qsort(files, *file_count, MAX_PATH_LENGTH, compare_strings);
    
    return 0;
}

/* Get image information from first file */
static int get_image_info(const char *dir_path, const char *filename, ImageInfo *info) {
    TIFF *tif;
    char full_path[MAX_PATH_LENGTH];
    uint32 width, height;
    uint16 bits_per_sample, samples_per_pixel, sample_format;
    
    snprintf(full_path, MAX_PATH_LENGTH, "%s%s%s", dir_path, PATH_SEPARATOR, filename);
    tif = TIFFOpen(full_path, "r");
    if (tif == NULL) {
        return -1;
    }
    
    TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &width);
    TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &height);
    TIFFGetField(tif, TIFFTAG_BITSPERSAMPLE, &bits_per_sample);
    TIFFGetField(tif, TIFFTAG_SAMPLESPERPIXEL, &samples_per_pixel);
    
    sample_format = SAMPLEFORMAT_UINT;
    TIFFGetFieldDefaulted(tif, TIFFTAG_SAMPLEFORMAT, &sample_format);
    
    TIFFClose(tif);
    
    info->width = width;
    info->height = height;
    info->bits_per_sample = bits_per_sample;
    info->samples_per_pixel = samples_per_pixel;
    info->sample_format = sample_format;
    info->bytes_per_pixel = (bits_per_sample / 8) * samples_per_pixel;
    info->bytes_per_slice = (size_t)width * height * info->bytes_per_pixel;
    
    return 0;
}

/* Get available GPU memory */
static size_t get_gpu_memory_available(void) {
    size_t free_mem, total_mem;
    
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    
    /* Use only a fraction of available memory */
    free_mem = (size_t)(free_mem * GPU_MEMORY_FRACTION);
    
    return free_mem / (1024 * 1024); /* Return in MB */
}

/* Calculate optimal GPU chunk size */
static int calculate_gpu_chunk_size(ImageInfo *info, int kernel_size) {
    size_t available_mb = get_gpu_memory_available();
    size_t slice_mb = info->bytes_per_slice / (1024 * 1024);
    int half_kernel = kernel_size / 2;
    int chunk_depth;
    
    /* Need memory for: input chunk, output chunk, and overlap buffers */
    /* Factor of 3 for safety (input, output, working memory) */
    chunk_depth = (int)(available_mb / (3 * slice_mb));
    
    /* Ensure minimum chunk size */
    if (chunk_depth < kernel_size + 2) {
        chunk_depth = kernel_size + 2;
    }
    
    /* Limit maximum chunk size for better overlap handling */
    if (chunk_depth > 128) {
        chunk_depth = 128;
    }
    
    log_message("GPU available memory: %zu MB, Slice size: %zu MB, GPU chunk depth: %d", 
                available_mb, slice_mb, chunk_depth);
    
    return chunk_depth;
}

/* Allocate GPU chunk with streams */
static GPUChunk* allocate_gpu_chunk(ImageInfo *info, int chunk_depth) {
    GPUChunk *chunk;
    size_t chunk_size;
    
    chunk = (GPUChunk*)malloc(sizeof(GPUChunk));
    if (chunk == NULL) {
        return NULL;
    }
    
    chunk->chunk_depth = chunk_depth;
    chunk_size = (size_t)chunk_depth * info->bytes_per_slice;
    
    /* Allocate pinned host memory for async transfers */
    CUDA_CHECK(hipHostMalloc(&chunk->h_data, chunk_size));
    
    /* Allocate device memory */
    CUDA_CHECK(hipMalloc(&chunk->d_data, chunk_size));
    CUDA_CHECK(hipMalloc(&chunk->d_output, chunk_size));
    
    /* Create stream for async operations */
    CUDA_CHECK(hipStreamCreate(&chunk->stream));
    
    return chunk;
}

/* Free GPU chunk */
static void free_gpu_chunk(GPUChunk *chunk) {
    if (chunk != NULL) {
        if (chunk->h_data != NULL) hipHostFree(chunk->h_data);
        if (chunk->d_data != NULL) hipFree(chunk->d_data);
        if (chunk->d_output != NULL) hipFree(chunk->d_output);
        if (chunk->stream != NULL) hipStreamDestroy(chunk->stream);
        free(chunk);
    }
}

/* Load chunk from disk to pinned host memory */
static int load_chunk_to_host(const char *dir_path, char files[][MAX_PATH_LENGTH], 
                              ImageInfo *info, GPUChunk *chunk) {
    TIFF *tif;
    char full_path[MAX_PATH_LENGTH];
    int z, y;
    tsize_t scanline_size;
    unsigned char *buffer;
    void *slice_data;
    
    for (z = 0; z < chunk->chunk_depth && (chunk->start_z + z) < info->depth; z++) {
        snprintf(full_path, MAX_PATH_LENGTH, "%s%s%s", dir_path, PATH_SEPARATOR, 
                 files[chunk->start_z + z]);
        tif = TIFFOpen(full_path, "r");
        if (tif == NULL) {
            log_message("Error: Cannot open %s", full_path);
            return -1;
        }
        
        scanline_size = TIFFScanlineSize(tif);
        buffer = (unsigned char*)_TIFFmalloc(scanline_size);
        if (buffer == NULL) {
            TIFFClose(tif);
            return -1;
        }
        
        slice_data = (char*)chunk->h_data + z * info->bytes_per_slice;
        
        for (y = 0; y < info->height; y++) {
            if (TIFFReadScanline(tif, buffer, y, 0) < 0) {
                _TIFFfree(buffer);
                TIFFClose(tif);
                return -1;
            }
            memcpy((char*)slice_data + y * scanline_size, buffer, scanline_size);
        }
        
        _TIFFfree(buffer);
        TIFFClose(tif);
    }
    
    chunk->end_z = chunk->start_z + z - 1;
    
    return 0;
}

/* Save chunk from pinned host memory to disk */
static int save_chunk_from_host(const char *dir_path, char files[][MAX_PATH_LENGTH], 
                                ImageInfo *info, GPUChunk *chunk) {
    TIFF *tif;
    char full_path[MAX_PATH_LENGTH];
    int z, y;
    tsize_t scanline_size;
    void *slice_data;
    
    for (z = 0; z < chunk->chunk_depth && (chunk->start_z + z) < info->depth; z++) {
        snprintf(full_path, MAX_PATH_LENGTH, "%s%s%s", dir_path, PATH_SEPARATOR, 
                 files[chunk->start_z + z]);
        tif = TIFFOpen(full_path, "w");
        if (tif == NULL) {
            log_message("Error: Cannot create %s", full_path);
            return -1;
        }
        
        /* Set TIFF tags */
        TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, info->width);
        TIFFSetField(tif, TIFFTAG_IMAGELENGTH, info->height);
        TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE, info->bits_per_sample);
        TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, info->samples_per_pixel);
        
        if (info->sample_format == SAMPLEFORMAT_UINT || 
            info->sample_format == SAMPLEFORMAT_INT ||
            info->sample_format == SAMPLEFORMAT_IEEEFP ||
            info->sample_format == SAMPLEFORMAT_VOID ||
            info->sample_format == SAMPLEFORMAT_COMPLEXINT ||
            info->sample_format == SAMPLEFORMAT_COMPLEXIEEEFP) {
            TIFFSetField(tif, TIFFTAG_SAMPLEFORMAT, info->sample_format);
        } else {
            if (info->bits_per_sample == 32) {
                TIFFSetField(tif, TIFFTAG_SAMPLEFORMAT, SAMPLEFORMAT_IEEEFP);
            } else {
                TIFFSetField(tif, TIFFTAG_SAMPLEFORMAT, SAMPLEFORMAT_UINT);
            }
        }
        
        TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
        TIFFSetField(tif, TIFFTAG_PHOTOMETRIC, PHOTOMETRIC_MINISBLACK);
        TIFFSetField(tif, TIFFTAG_COMPRESSION, COMPRESSION_NONE);
        
        slice_data = (char*)chunk->h_data + z * info->bytes_per_slice;
        scanline_size = info->width * info->bytes_per_pixel;
        
        for (y = 0; y < info->height; y++) {
            if (TIFFWriteScanline(tif, (char*)slice_data + y * scanline_size, y, 0) < 0) {
                TIFFClose(tif);
                return -1;
            }
        }
        
        TIFFClose(tif);
    }
    
    return 0;
}

/* Process chunk on GPU with overlap handling */
static void process_chunk_on_gpu(GPUChunk *chunk, ImageInfo *info, FilterParams *params,
                                GPUChunk *prev_chunk, GPUChunk *next_chunk) {
    dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 grid(
        (info->width + block.x - 1) / block.x,
        (info->height + block.y - 1) / block.y,
        (chunk->chunk_depth + block.z - 1) / block.z
    );
    
    float spatial_sigma_sq_inv = 0.5f / (params->spatial_sigma * params->spatial_sigma);
    float intensity_sigma_sq_inv = 0.5f / (params->intensity_sigma * params->intensity_sigma);
    float max_value;
    
    /* Set max value based on bit depth */
    if (info->bits_per_sample == 8) {
        max_value = 255.0f;
    } else if (info->bits_per_sample == 16) {
        max_value = 65535.0f;
    } else if (info->bits_per_sample == 32) {
        if (info->sample_format == SAMPLEFORMAT_IEEEFP) {
            max_value = FLT_MAX;
        } else {
            max_value = 4294967295.0f;
        }
    } else {
        max_value = FLT_MAX;
    }
    
    /* Prepare overlap data pointers */
    void *d_prev_overlap = NULL;
    void *d_next_overlap = NULL;
    int prev_overlap_depth = 0;
    int next_overlap_depth = 0;
    int half_kernel = params->kernel_size / 2;
    
    /* Copy overlap regions if available */
    if (prev_chunk != NULL && chunk->start_z > 0) {
        prev_overlap_depth = half_kernel;
        size_t overlap_offset = (prev_chunk->chunk_depth - prev_overlap_depth) * info->bytes_per_slice;
        d_prev_overlap = (char*)prev_chunk->d_output + overlap_offset;
    }
    
    if (next_chunk != NULL && chunk->end_z < info->depth - 1) {
        next_overlap_depth = half_kernel;
        d_next_overlap = next_chunk->d_data;
    }
    
    /* Launch kernel based on data type */
    if (info->bits_per_sample == 8) {
        bilateral_filter_3d_kernel_stream<unsigned char><<<grid, block, 0, chunk->stream>>>(
            (unsigned char*)chunk->d_data, (unsigned char*)chunk->d_output,
            info->width, info->height, chunk->chunk_depth,
            params->kernel_size, spatial_sigma_sq_inv, intensity_sigma_sq_inv, max_value,
            chunk->start_z, info->depth,
            (unsigned char*)d_prev_overlap, (unsigned char*)d_next_overlap,
            prev_overlap_depth, next_overlap_depth
        );
    } else if (info->bits_per_sample == 16) {
        bilateral_filter_3d_kernel_stream<unsigned short><<<grid, block, 0, chunk->stream>>>(
            (unsigned short*)chunk->d_data, (unsigned short*)chunk->d_output,
            info->width, info->height, chunk->chunk_depth,
            params->kernel_size, spatial_sigma_sq_inv, intensity_sigma_sq_inv, max_value,
            chunk->start_z, info->depth,
            (unsigned short*)d_prev_overlap, (unsigned short*)d_next_overlap,
            prev_overlap_depth, next_overlap_depth
        );
    } else if (info->bits_per_sample == 32 && info->sample_format == SAMPLEFORMAT_IEEEFP) {
        bilateral_filter_3d_kernel_stream<float><<<grid, block, 0, chunk->stream>>>(
            (float*)chunk->d_data, (float*)chunk->d_output,
            info->width, info->height, chunk->chunk_depth,
            params->kernel_size, spatial_sigma_sq_inv, intensity_sigma_sq_inv, max_value,
            chunk->start_z, info->depth,
            (float*)d_prev_overlap, (float*)d_next_overlap,
            prev_overlap_depth, next_overlap_depth
        );
    }
    
    /* Check for kernel errors */
    CUDA_CHECK(hipGetLastError());
}

/* Get current time in seconds */
static double get_current_time(void) {
#ifdef _WIN32
    LARGE_INTEGER frequency, counter;
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&counter);
    return (double)counter.QuadPart / (double)frequency.QuadPart;
#else
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec / 1e9;
#endif
}

/* Get dynamic range for 32-bit float images by sampling */
static float get_float_dynamic_range(const char *dir_path, char files[][MAX_PATH_LENGTH], 
                                    int file_count, ImageInfo *info) {
    TIFF *tif;
    char full_path[MAX_PATH_LENGTH];
    tsize_t scanline_size;
    float *buffer;
    int sample_count, sample_interval;
    int i, y, x;
    float min_val = FLT_MAX;
    float max_val = -FLT_MAX;
    
    /* Sample 1/20 of images */
    sample_count = file_count / 20;
    if (sample_count < 1) sample_count = 1;
    if (sample_count > 10) sample_count = 10;  /* Limit to 10 samples */
    
    sample_interval = file_count / sample_count;
    
    log_message("Sampling %d images to determine dynamic range...", sample_count);
    
    for (i = 0; i < sample_count; i++) {
        int file_idx = i * sample_interval;
        snprintf(full_path, MAX_PATH_LENGTH, "%s%s%s", dir_path, PATH_SEPARATOR, files[file_idx]);
        
        tif = TIFFOpen(full_path, "r");
        if (tif == NULL) continue;
        
        scanline_size = TIFFScanlineSize(tif);
        buffer = (float*)_TIFFmalloc(scanline_size);
        if (buffer == NULL) {
            TIFFClose(tif);
            continue;
        }
        
        /* Sample center region of image */
        int start_y = info->height / 4;
        int end_y = 3 * info->height / 4;
        int y_step = (end_y - start_y) / 10;  /* Sample 10 lines per image */
        if (y_step < 1) y_step = 1;
        
        for (y = start_y; y < end_y; y += y_step) {
            if (TIFFReadScanline(tif, buffer, y, 0) >= 0) {
                int start_x = info->width / 4;
                int end_x = 3 * info->width / 4;
                for (x = start_x; x < end_x; x++) {
                    float val = buffer[x];
                    if (isfinite(val)) {  /* Skip NaN and Inf */
                        if (val < min_val) min_val = val;
                        if (val > max_val) max_val = val;
                    }
                }
            }
        }
        
        _TIFFfree(buffer);
        TIFFClose(tif);
    }
    
    float range = max_val - min_val;
    log_message("Detected dynamic range: [%.3f, %.3f], range: %.3f", 
                min_val, max_val, range);
    
    return range;
}

/* Main function */
int main(int argc, char *argv[]) {
    char input_dir[MAX_PATH_LENGTH];
    char output_dir[MAX_PATH_LENGTH];
    FilterParams params;
    char (*files)[MAX_PATH_LENGTH];
    int file_count;
    ImageInfo info;
    GPUChunk *chunks[3];  /* Triple buffering: processing, loading, saving */
    int chunk_size, num_chunks, chunk_idx;
    int half_kernel;
    double start_time, end_time, total_start_time;
    size_t total_size_mb, chunk_bytes;
    int current_chunk, next_chunk, prev_chunk;
    
    int	i;
    
    /* Initialize CUDA */
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    if (device_count == 0) {
        fprintf(stderr, "Error: No CUDA-capable devices found\n");
        return 1;
    }
    
    /* Select best device */
    int best_device = 0;
    int max_sm_count = 0;
    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        int sm_count = prop.multiProcessorCount;
        if (sm_count > max_sm_count) {
            max_sm_count = sm_count;
            best_device = i;
        }
    }
    CUDA_CHECK(hipSetDevice(best_device));
    
    /* Initialize logging */
    init_logging(argv[0]);
    
    /* Parse command line arguments */
    if (argc < 3) {
        printf("Usage: %s <input_dir> <output_dir> [kernel_size] [spatial_sigma] "
                    "[intensity_sigma]", argv[0]);
//        close_logging();
        return 1;
    }
    
    strncpy(input_dir, argv[1], MAX_PATH_LENGTH - 1);
    input_dir[MAX_PATH_LENGTH - 1] = '\0';
    strncpy(output_dir, argv[2], MAX_PATH_LENGTH - 1);
    output_dir[MAX_PATH_LENGTH - 1] = '\0';
    
    /* Set default parameters */
    params.kernel_size = 5;
    params.spatial_sigma = 2.0f;
    params.intensity_sigma = 50.0f;
    
    /* Parse optional parameters */
    if (argc > 3) params.kernel_size = atoi(argv[3]);
    if (argc > 4) params.spatial_sigma = (float)atof(argv[4]);
    if (argc > 5) params.intensity_sigma = (float)atof(argv[5]);
    
    /* Validate parameters */
    if (params.kernel_size < 3 || params.kernel_size > MAX_KERNEL_SIZE || params.kernel_size % 2 == 0) {
        log_message("Error: Kernel size must be odd and between 3 and %d", MAX_KERNEL_SIZE);
        close_logging();
        return 1;
    }
    
    if (params.spatial_sigma <= 0 || params.intensity_sigma <= 0) {
        log_message("Error: Sigma values must be positive");
        close_logging();
        return 1;
    }
    
    half_kernel = params.kernel_size / 2;
    
    log_message("Input directory: %s", input_dir);
    log_message("Output directory: %s", output_dir);
    
    /* Create output directory */
    if (create_directory(output_dir) != 0) {
        log_message("Error: Cannot create output directory");
        close_logging();
        return 1;
    }
    
    /* Allocate file list */
    files = (char (*)[MAX_PATH_LENGTH])malloc(MAX_FILES * MAX_PATH_LENGTH);
    if (files == NULL) {
        log_message("Error: Memory allocation failed for file list");
        close_logging();
        return 1;
    }
    
    /* Get list of TIFF files */
    if (get_tiff_files(input_dir, files, &file_count) != 0) {
        log_message("Error: Cannot read input directory");
        free(files);
        close_logging();
        return 1;
    }
    
    if (file_count == 0) {
        log_message("Error: No TIFF files found in input directory");
        free(files);
        close_logging();
        return 1;
    }
    
    log_message("Found %d TIFF files", file_count);
    
    /* Get image information */
    if (get_image_info(input_dir, files[0], &info) != 0) {
        log_message("Error: Cannot read image information");
        free(files);
        close_logging();
        return 1;
    }
    
    info.depth = file_count;
    total_size_mb = (size_t)info.bytes_per_slice * file_count / (1024 * 1024);
    
    log_message("Image dimensions: %ux%ux%u", info.width, info.height, info.depth);
    log_message("Bits per sample: %u", info.bits_per_sample);
    log_message("Sample format: %u", info.sample_format);
    log_message("Total data size: %zu MB", total_size_mb);
    
    /* Auto-adjust intensity_sigma based on bit depth if using default value */
    if (argc <= 5 && params.intensity_sigma == 50.0f) {
        if (info.bits_per_sample == 8) {
            params.intensity_sigma = 256.0f / 3.0f;  /* ~85 */
            log_message("Auto-adjusted intensity_sigma to %.1f for 8-bit images", 
                        params.intensity_sigma);
        } else if (info.bits_per_sample == 16) {
            params.intensity_sigma = 65536.0f / 3.0f;  /* ~21845 */
            log_message("Auto-adjusted intensity_sigma to %.1f for 16-bit images", 
                        params.intensity_sigma);
        } else if (info.bits_per_sample == 32 && info.sample_format == SAMPLEFORMAT_IEEEFP) {
            /* For 32-bit float, sample images to determine dynamic range */
            float dynamic_range = get_float_dynamic_range(input_dir, files, file_count, &info);
            if (dynamic_range > 0.0f) {
                params.intensity_sigma = dynamic_range / 3.0f;
                log_message("Auto-adjusted intensity_sigma to %.3f for 32-bit float images", 
                            params.intensity_sigma);
            } else {
                params.intensity_sigma = 0.1f;  /* Fallback for normalized 0-1 range */
                log_message("Using default intensity_sigma %.3f for 32-bit float images", 
                            params.intensity_sigma);
            }
        }
    }
    
    /* Calculate chunk size */
    chunk_size = calculate_gpu_chunk_size(&info, params.kernel_size);
    num_chunks = (file_count + chunk_size - 1) / chunk_size;
    chunk_bytes = (size_t)chunk_size * info.bytes_per_slice;
    
    log_message("Processing in %d chunks of %d slices each", num_chunks, chunk_size);
    log_message("Chunk size: %.2f MB", chunk_bytes / (1024.0 * 1024.0));
    log_message("Kernel size: %d", params.kernel_size);
    log_message("Spatial sigma: %.2f", params.spatial_sigma);
    log_message("Intensity sigma: %.2f", params.intensity_sigma);
    
    total_start_time = get_current_time();
    
    /* Initialize chunks for triple buffering */
    for (int i = 0; i < 3; i++) {
        chunks[i] = allocate_gpu_chunk(&info, chunk_size + 2 * half_kernel);
        if (chunks[i] == NULL) {
            log_message("Error: Failed to allocate GPU chunk %d", i);
            for (int j = 0; j < i; j++) {
                free_gpu_chunk(chunks[j]);
            }
            free(files);
            close_logging();
            return 1;
        }
    }
    
    /* Process chunks with triple buffering */
    current_chunk = 0;
    next_chunk = 1;
    prev_chunk = 2;
    
    for (chunk_idx = 0; chunk_idx < num_chunks; chunk_idx++) {
        int chunk_start, chunk_end, actual_chunk_depth;
        int overlap_start, overlap_end;
        
        log_message("Processing chunk %d/%d", chunk_idx + 1, num_chunks);
        
        /* Calculate chunk boundaries */
        chunk_start = chunk_idx * chunk_size;
        chunk_end = chunk_start + chunk_size - 1;
        if (chunk_end >= file_count) {
            chunk_end = file_count - 1;
        }
        
        /* Add overlap for kernel */
        overlap_start = chunk_start - half_kernel;
        overlap_end = chunk_end + half_kernel;
        
        if (overlap_start < 0) overlap_start = 0;
        if (overlap_end >= file_count) overlap_end = file_count - 1;
        
        actual_chunk_depth = overlap_end - overlap_start + 1;
        
        /* Set chunk parameters */
        chunks[current_chunk]->start_z = overlap_start;
        chunks[current_chunk]->chunk_depth = actual_chunk_depth;
        
        /* Load current chunk */
        start_time = get_current_time();
        if (load_chunk_to_host(input_dir, files, &info, chunks[current_chunk]) != 0) {
            log_message("Error: Failed to load chunk");
            for (int i = 0; i < 3; i++) {
                free_gpu_chunk(chunks[i]);
            }
            free(files);
            close_logging();
            return 1;
        }
        end_time = get_current_time();
        log_message("Loaded chunk in %.2f seconds", end_time - start_time);
        
        /* Transfer to GPU */
        start_time = get_current_time();
        CUDA_CHECK(hipMemcpyAsync(chunks[current_chunk]->d_data, chunks[current_chunk]->h_data,
                                   chunk_bytes, hipMemcpyHostToDevice, chunks[current_chunk]->stream));
        
        /* Process on GPU */
        process_chunk_on_gpu(chunks[current_chunk], &info, &params,
                            (chunk_idx > 0) ? chunks[prev_chunk] : NULL,
                            (chunk_idx < num_chunks - 1) ? chunks[next_chunk] : NULL);
        
        /* Transfer back to host */
        CUDA_CHECK(hipMemcpyAsync(chunks[current_chunk]->h_data, chunks[current_chunk]->d_output,
                                   chunk_bytes, hipMemcpyDeviceToHost, chunks[current_chunk]->stream));
        
        /* Synchronize and measure GPU time */
        CUDA_CHECK(hipStreamSynchronize(chunks[current_chunk]->stream));
        end_time = get_current_time();
        log_message("GPU processing in %.2f seconds", end_time - start_time);
        
        /* Save chunk */
        start_time = get_current_time();
        if (save_chunk_from_host(output_dir, files, &info, chunks[current_chunk]) != 0) {
            log_message("Error: Failed to save chunk");
            for (int i = 0; i < 3; i++) {
                free_gpu_chunk(chunks[i]);
            }
            free(files);
            close_logging();
            return 1;
        }
        end_time = get_current_time();
        log_message("Saved chunk in %.2f seconds", end_time - start_time);
        
        /* Rotate chunk indices for triple buffering */
        int temp = prev_chunk;
        prev_chunk = current_chunk;
        current_chunk = next_chunk;
        next_chunk = temp;
    }
    
    /* Clean up */
    for (int i = 0; i < 3; i++) {
        free_gpu_chunk(chunks[i]);
    }
    free(files);
    
    /* Report total processing time */
    end_time = get_current_time();
    log_message("Total processing time: %.2f seconds", end_time - total_start_time);
    
    /* Calculate and log performance metrics */
    size_t total_pixels = (size_t)info.width * info.height * info.depth;
    double mpixels_per_second = total_pixels / ((end_time - total_start_time) * 1e6);
    log_message("Performance: %.2f MPixels/second", mpixels_per_second);
    
    log_message("Processing completed successfully");
    close_logging();
    
    /* Reset CUDA device */
    CUDA_CHECK(hipDeviceReset());
    
// append to log file
	FILE		*f;
	if ((f = fopen("cmd-hst.log", "a")) == NULL) {
		return(-1);
	}
	for (i = 0; i<argc; ++i) fprintf(f, "%s ", argv[i]);
    fprintf(f,"  %% Kernel size: %d", params.kernel_size);
    fprintf(f," Spatial sigma: %.2f", params.spatial_sigma);
    fprintf(f," Intensity sigma: %.2f", params.intensity_sigma);
	fprintf(f, "\n");
	fclose(f);

    return 0;
}